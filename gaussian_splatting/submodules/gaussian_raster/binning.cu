#include "hip/hip_runtime.h"
#ifndef __HIPCC__
    #define __HIPCC__
    #define __NVCC__
#endif
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cuda/atomic>
namespace cg = cooperative_groups;

#include <c10/cuda/CUDAException.h>
#include <ATen/core/TensorAccessor.h>

#include "cuda_errchk.h"
#include "binning.h"

 __global__ void duplicate_with_keys_kernel(
    const torch::PackedTensorAccessor32<int32_t, 3,torch::RestrictPtrTraits> LU,//viewnum,2,pointnum
    const torch::PackedTensorAccessor32<int32_t, 3,torch::RestrictPtrTraits> RD,//viewnum,2,pointnum
    const torch::PackedTensorAccessor32<int32_t, 2,torch::RestrictPtrTraits> prefix_sum,//viewnum,pointnum
     const torch::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits> depth_sorted_pointid,//viewnum,pointnum
    int TileSizeX,
    torch::PackedTensorAccessor32 < int16_t, 2, torch::RestrictPtrTraits> table_tileId,
     torch::PackedTensorAccessor32 < int32_t, 2, torch::RestrictPtrTraits> table_pointId
    )
{
    int view_id = blockIdx.y;
    

    if (blockIdx.x * blockDim.x + threadIdx.x < prefix_sum.size(1))
    {
        int point_id = depth_sorted_pointid[view_id][blockIdx.x * blockDim.x + threadIdx.x];
        int end = prefix_sum[view_id][blockIdx.x * blockDim.x + threadIdx.x];

        //int end = prefix_sum[view_id][point_id+1];
        int l = LU[view_id][0][point_id];
        int u = LU[view_id][1][point_id];
        int r = RD[view_id][0][point_id];
        int d = RD[view_id][1][point_id];
        int count = 0;

        for (int i = u; i < d; i++)
        {
            for (int j = l; j < r; j++)
            {
                int tile_id = i * TileSizeX + j;
                table_tileId[view_id][end - 1 - count] = tile_id+1;// tile_id 0 means invalid!
                table_pointId[view_id][end - 1 - count] = point_id;
                count++;
            }
        }
    }


}



std::vector<at::Tensor> duplicateWithKeys(at::Tensor LU, at::Tensor RD, at::Tensor prefix_sum, at::Tensor depth_sorted_pointid, int64_t allocate_size, int64_t TilesSizeX)
{
    at::DeviceGuard guard(LU.device());
    int64_t view_num = LU.sizes()[0];
    int64_t points_num = LU.sizes()[2];

    std::vector<int64_t> output_shape{ view_num, allocate_size };

    auto opt = torch::TensorOptions().dtype(torch::kInt16).layout(torch::kStrided).device(LU.device()).requires_grad(false);
    auto table_tileId = torch::zeros(output_shape, opt);
    opt = torch::TensorOptions().dtype(torch::kInt32).layout(torch::kStrided).device(LU.device()).requires_grad(false);
    auto table_pointId= torch::zeros(output_shape, opt);

    dim3 Block3d(std::ceil(points_num/1024.0f), view_num, 1);
    

    duplicate_with_keys_kernel<<<Block3d ,1024>>>(
        LU.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        RD.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
        prefix_sum.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        depth_sorted_pointid.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>(),
        TilesSizeX,
        table_tileId.packed_accessor32<int16_t, 2, torch::RestrictPtrTraits>(),
        table_pointId.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
    

    return { table_tileId ,table_pointId };
    
}

__global__ void tile_range_kernel(
    const torch::PackedTensorAccessor32<int16_t, 2,torch::RestrictPtrTraits> table_tileId,//viewnum,pointnum
    int table_length,
    int max_tileId,
    torch::PackedTensorAccessor32 < int32_t, 2, torch::RestrictPtrTraits> tile_range
)
{
    int view_id = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;


    // head
    if (index == 0)
    {
        int tile_id=table_tileId[view_id][index];
        tile_range[view_id][tile_id] = index;
    }
    
    //tail
    if (index == table_length - 1)
    {
        tile_range[view_id][max_tileId + 1] = table_length;
    }
    
    if (index < table_length-1)
    {
        int cur_tile = table_tileId[view_id][index];
        int next_tile= table_tileId[view_id][index+1];
        if (cur_tile!=next_tile)
        {
            if (cur_tile + 1 < next_tile)
            {
                tile_range[view_id][cur_tile + 1] = index + 1;
            }
            tile_range[view_id][next_tile] = index + 1;
        }
    }
}

at::Tensor tileRange(at::Tensor table_tileId, int64_t table_length, int64_t max_tileId)
{
    at::DeviceGuard guard(table_tileId.device());

    int64_t view_num = table_tileId.sizes()[0];
    std::vector<int64_t> output_shape{ view_num,max_tileId + 1 + 1 };//+1 for tail
    //printf("\ntensor shape in tileRange:%ld,%ld\n", view_num, max_tileId+1-1);
    auto opt = torch::TensorOptions().dtype(torch::kInt32).layout(torch::kStrided).device(table_tileId.device()).requires_grad(false);
    auto out = torch::ones(output_shape, opt)*-1;

    dim3 Block3d(std::ceil(table_length / 1024.0f), view_num, 1);

    tile_range_kernel<<<Block3d, 1024 >>>
        (table_tileId.packed_accessor32<int16_t, 2, torch::RestrictPtrTraits>(), table_length, max_tileId, out.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;

    return out;
}


