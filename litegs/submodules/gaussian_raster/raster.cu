#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#define __NVCC__
#endif
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cuda/atomic>
#include <math.h>
#include <hip/hip_fp16.h>
namespace cg = cooperative_groups;

#include <c10/cuda/CUDAException.h>
#include <ATen/core/TensorAccessor.h>

#include "cuda_errchk.h"
#include "raster.h"


struct PackedParams
{
    float ndc_x;
    float ndc_y;
    float ndc_z;
    float inv_cov00;
    float inv_cov01;
    float inv_cov11;
};

struct RGBA16
{
    half r;
    half g;
    half b;
    half a;
};

struct RGBA32
{
    float r;
    float g;
    float b;
    float a;
};

struct RGBA16x2
{
    half2 r;
    half2 g;
    half2 b;
    half2 a;
};

struct RegisterBuffer
{
    half2 r;
    half2 g;
    half2 b;
    half2 t;
    unsigned int lst_contributor;//simd ushort2
    half2 alpha;
};

#define __HALF2_TO_UI(var) *(reinterpret_cast<unsigned int *>(&(var)))
#define __HALF2_TO_CUI(var) *(reinterpret_cast<const unsigned int *>(&(var)))
inline __device__ half2 fast_exp_approx(half2 input) {
    half2 output;
    half2 log2_e(1.4426950409f, 1.4426950409f);
    half2 scaled_input = input * log2_e;
    asm("ex2.approx.f16x2 %0, %1;" : "=r"(__HALF2_TO_UI(output)) : "r"(__HALF2_TO_CUI(scaled_input)));
    return output;
}

template <int tile_size_y, int tile_size_x, bool enable_trans, bool enable_depth>
__global__ void raster_forward_kernel(
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> sorted_points,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> start_index,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<float/*torch::Half*/, 3, torch::RestrictPtrTraits> packed_params,         //[batch,point_num,6]
    const torch::PackedTensorAccessor32<torch::Half, 3, torch::RestrictPtrTraits> packed_rgba16,         //[batch,point_num,4]
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> specific_tiles,          //[batch,tiles_num]
    torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> output_img,    //[batch,3,tile,tilesize,tilesize]
    torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> output_transmitance,    //[batch,1,tile,tilesize,tilesize]
    torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> output_depth,     //[batch,1,tile,tilesize, tilesize]
    torch::PackedTensorAccessor32<short, 4, torch::RestrictPtrTraits> output_last_contributor,    //[batch,tile,tilesize,tilesize]
    int tiles_num_x, int img_h, int img_w
)
{
    //assert blockDim.x==32

    constexpr int VECTOR_SIZE = 2;
    constexpr int PIXELS_PER_THREAD = (tile_size_x * tile_size_y) / (32* VECTOR_SIZE);//half2: 32 pixel per warp->64 pixel per warp
    constexpr float SCALER = 128.0f;
    constexpr float INV_SCALER = 1.0f / 128;

    const int batch_id = blockIdx.y;
    int tile_id = blockIdx.x * blockDim.y + threadIdx.y + 1;// +1, tile_id 0 is invalid
    if (specific_tiles.size(1) != 0 && (blockIdx.x * blockDim.y + threadIdx.y < specific_tiles.size(1)))
    {
        tile_id = specific_tiles[batch_id][blockIdx.x * blockDim.y + threadIdx.y];
    }

    if (tile_id != 0 && tile_id < start_index.size(1) - 1)
    {

        int start_index_in_tile = start_index[batch_id][tile_id];
        int end_index_in_tile = start_index[batch_id][tile_id + 1];

        if (start_index_in_tile != -1)
        {
            RegisterBuffer reg_buffer[PIXELS_PER_THREAD];
#pragma unroll
            for (int i = 0; i < PIXELS_PER_THREAD; i++)
            {
                reg_buffer[i].r = half2(0, 0);
                reg_buffer[i].g = half2(0, 0);
                reg_buffer[i].b = half2(0, 0);
                //alpha_min 1/256
                //t_min 1/8192
                //-> t_mul_alpha_min 1/(256*8192) -> half underflow
                reg_buffer[i].t = half2(SCALER, SCALER);//mul 128.0f to avoid underflow; t_max * 128 * color_max < half_max;
                reg_buffer[i].lst_contributor = 0;//simd ushort2
            }

            unsigned int any_active = 0xffffffffu;
            int index_in_tile = 0;
            auto points_id_in_tile = &sorted_points[batch_id][start_index_in_tile];
            for (; (index_in_tile+ start_index_in_tile < end_index_in_tile) && (any_active != 0); index_in_tile++)
            {
                int point_id = points_id_in_tile[index_in_tile];
                PackedParams params = *((PackedParams*)&packed_params[batch_id][point_id][0]);
                RGBA16 temp = *((RGBA16*)&packed_rgba16[batch_id][point_id][0]);
                RGBA16x2 point_color_x2;
                point_color_x2.r = half2(temp.r, temp.r);
                point_color_x2.g = half2(temp.g, temp.g);
                point_color_x2.b = half2(temp.b, temp.b);
                point_color_x2.a = half2(temp.a, temp.a);
                float2 xy{ (params.ndc_x + 1.0f) * 0.5f * img_w - 0.5f ,(params.ndc_y + 1.0f) * 0.5f * img_h - 0.5f };

                const int pixel_x = ((tile_id - 1) % tiles_num_x) * tile_size_x + threadIdx.x % tile_size_x ;
                const int pixel_y = ((tile_id - 1) / tiles_num_x) * tile_size_y + threadIdx.x / tile_size_x * PIXELS_PER_THREAD * VECTOR_SIZE;
                float2 d { xy.x - pixel_x,xy.y - pixel_y };
                float basic = -0.5f * (params.inv_cov00 * d.x * d.x + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * d.x * d.y);
                float bxcy = params.inv_cov11 * d.y + params.inv_cov01 * d.x;
                float neg_half_c = -0.5f * params.inv_cov11;
                //basic+=(cy+bx)*delta - 0.5*c*delta*delta

                any_active = 0;
#pragma unroll
                for (int i = 0; i < PIXELS_PER_THREAD; i++)
                {
                    half2 power{
                        basic + 2 * i * bxcy + 2 * i * 2 * i * neg_half_c,
                        basic + (2 * i + 1) * bxcy + (2 * i + 1) * (2 * i + 1) * neg_half_c
                    };
                    unsigned int active_mask = 0xffffffffu;
                    active_mask = __hgt2_mask(reg_buffer[i].t, half2(SCALER / 8192, SCALER / 8192));
                    any_active |= active_mask;

                    unsigned int alpha_valid_mask = 0xffffffffu;
                    alpha_valid_mask &= __hle2_mask(power, half2(0, 0));
                    reg_buffer[i].alpha = point_color_x2.a * fast_exp_approx(power);
                    alpha_valid_mask &= __hge2_mask(reg_buffer[i].alpha, half2(1.0f / 256, 1.0f / 256));
                    reg_buffer[i].alpha = __hmin2(half2(255.0f / 256, 255.0f / 256), reg_buffer[i].alpha);

                    reg_buffer[i].lst_contributor += (0x00010001 & active_mask);
                    reinterpret_cast<unsigned int*>(&reg_buffer[i].alpha)[0] &= (active_mask & alpha_valid_mask);

                    half2 weight = reg_buffer[i].t * reg_buffer[i].alpha;
                    reg_buffer[i].r += (point_color_x2.r * weight);
                    reg_buffer[i].g += (point_color_x2.g * weight);
                    reg_buffer[i].b += (point_color_x2.b * weight);
                    reg_buffer[i].t = reg_buffer[i].t * (half2(1.0f, 1.0f) - reg_buffer[i].alpha);
                }
                //reg_buffer[1].alpha = (half2(2.0f, 2.0f) * reg_buffer[0].alpha + reg_buffer[3].alpha) * half2(1.0f / 3, 1.0f / 3);
                //reg_buffer[2].alpha = (reg_buffer[0].alpha + half2(2.0f, 2.0f) * reg_buffer[3].alpha) * half2(1.0f / 3, 1.0f / 3);

            }


            int tile_index = blockIdx.x * blockDim.y + threadIdx.y;
            auto ourput_r = output_img[batch_id][0][tile_index];
            auto ourput_g = output_img[batch_id][1][tile_index];
            auto ourput_b = output_img[batch_id][2][tile_index];
            auto ourput_t = output_transmitance[batch_id][0][tile_index];
            auto output_last_index = output_last_contributor[batch_id][tile_index];
#pragma unroll
            for (int i = 0; i < PIXELS_PER_THREAD; i++)
            {
                const int output_x = threadIdx.x % tile_size_x;
                const int output_y = threadIdx.x / tile_size_x * PIXELS_PER_THREAD * VECTOR_SIZE + 2 * i;

                ourput_r[output_y][output_x] = float(reg_buffer[i].r.x) * INV_SCALER;
                ourput_r[output_y + 1][output_x] = float(reg_buffer[i].r.y) * INV_SCALER;

                ourput_g[output_y][output_x] = float(reg_buffer[i].g.x) * INV_SCALER;
                ourput_g[output_y + 1][output_x] = float(reg_buffer[i].g.y) * INV_SCALER;

                ourput_b[output_y][output_x] = float(reg_buffer[i].b.x) * INV_SCALER;
                ourput_b[output_y + 1][output_x] = float(reg_buffer[i].b.y) * INV_SCALER;

                ourput_t[output_y][output_x] = float(reg_buffer[i].t.x) * INV_SCALER;
                ourput_t[output_y + 1][output_x] = float(reg_buffer[i].t.y) * INV_SCALER;

                output_last_index[output_y][output_x] = reg_buffer[i].lst_contributor&0xff;
                output_last_index[output_y + 1][output_x] = (reg_buffer[i].lst_contributor >> 16) & 0xff;
            }
        }
    }
}


std::vector<at::Tensor> rasterize_forward(
    at::Tensor sorted_points,
    at::Tensor start_index,
    at::Tensor  packed_params,//packed param
    at::Tensor  rgba16,//packed param
    std::optional<at::Tensor>  specific_tiles_arg,
    int64_t img_h,
    int64_t img_w,
    int64_t tile_h,
    int64_t tile_w,
    bool enable_trans,
    bool enable_depth
)
{
    at::DeviceGuard guard(packed_params.device());

    int64_t viewsnum = start_index.sizes()[0];
    int tilesnum_x = std::ceil(img_w / float(tile_w));
    int tilesnum_y = std::ceil(img_h / float(tile_h));
    int64_t tilesnum = tilesnum_x * tilesnum_y;
    at::Tensor specific_tiles;
    if (specific_tiles_arg.has_value())
    {
        specific_tiles = *specific_tiles_arg;
        tilesnum = specific_tiles.sizes()[1];
    }
    else
    {
        specific_tiles = torch::empty({ 0,0 }, packed_params.options().dtype(torch::kInt32));
    }


    torch::TensorOptions opt_img = torch::TensorOptions().dtype(torch::kFloat32).layout(torch::kStrided).device(start_index.device()).requires_grad(true);
    at::Tensor output_img = torch::empty({ viewsnum,3, tilesnum,tile_h,tile_w }, opt_img);

    torch::TensorOptions opt_t = torch::TensorOptions().dtype(torch::kFloat32).layout(torch::kStrided).device(start_index.device()).requires_grad(enable_trans);
    at::Tensor output_transmitance = torch::empty({ viewsnum,1, tilesnum, tile_h, tile_w }, opt_t);

    at::Tensor output_depth = torch::empty({ 0,0, 0, 0, 0 }, opt_t);
    if (enable_depth)
    {
        output_depth = torch::empty({ viewsnum,1, tilesnum, tile_h, tile_w }, opt_t.requires_grad(true));
    }

    torch::TensorOptions opt_c = torch::TensorOptions().dtype(torch::kShort).layout(torch::kStrided).device(start_index.device()).requires_grad(false);
    at::Tensor output_last_contributor = torch::empty({ viewsnum, tilesnum, tile_h, tile_w }, opt_c);


    int tiles_per_block = 4;
    dim3 Block3d(std::ceil(tilesnum / float(tiles_per_block)), viewsnum, 1);
    dim3 Thread3d(32, tiles_per_block);

    raster_forward_kernel<16, 16, false, false> << <Block3d, Thread3d >> > (sorted_points.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        start_index.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        packed_params.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        rgba16.packed_accessor32<torch::Half, 3, torch::RestrictPtrTraits>(),
        specific_tiles.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        output_img.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        output_transmitance.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        output_depth.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        output_last_contributor.packed_accessor32<short, 4, torch::RestrictPtrTraits>(),
        tilesnum_x, img_h, img_w);

    CUDA_CHECK_ERRORS;

    return { output_img ,output_transmitance,output_depth ,output_last_contributor };
}


struct BackwardRegisterBuffer
{
    half2 r;
    half2 g;
    half2 b;
    half2 t;
    half2 alpha;
};


template<class T, bool boardcast>
inline __device__ void warp_reduce_sum(T& data)
{
    data += __shfl_down_sync(0xffffffff, data, 16);
    data += __shfl_down_sync(0xffffffff, data, 8);
    data += __shfl_down_sync(0xffffffff, data, 4);
    data += __shfl_down_sync(0xffffffff, data, 2);
    data += __shfl_down_sync(0xffffffff, data, 1);
    if (boardcast)
        data = __shfl_sync(0xffffffff, data, 0);
}

template<>
inline __device__ void warp_reduce_sum<float, false>(float& data)
{
    int exponent = (__float_as_uint(data) >> 23) & 0xff;
    exponent = __reduce_max_sync(0xffffffff, exponent) - 127;
    int scale_exponent = 23 - exponent;
    bool valid = (exponent > -127) && (scale_exponent < 128);

    float scaler = __uint_as_float(0 | ((scale_exponent + 127) << 23));
    float inv_scaler = __uint_as_float(0 | ((127 - scale_exponent) << 23));
    int scaled_value = static_cast<int>(data * scaler);
    scaled_value = __reduce_add_sync(0xffffffff, scaled_value) * valid;

    data = scaled_value * inv_scaler;
}

template<>
inline __device__ void warp_reduce_sum<float2, false>(float2& data)
{
    int exponent = (__float_as_uint(data.x) >> 23) & 0xff;
    exponent = max(exponent, (__float_as_uint(data.y) >> 23) & 0xff);
    exponent = __reduce_max_sync(0xffffffff, exponent) - 127;
    int scale_exponent = 23 - exponent;
    bool valid = (exponent > -127) && (scale_exponent < 128);

    float scaler = __uint_as_float(0 | ((scale_exponent + 127) << 23));
    float inv_scaler = __uint_as_float(0 | ((127 - scale_exponent) << 23));

    int scaled_value_x = static_cast<int>(data.x * scaler);
    scaled_value_x = __reduce_add_sync(0xffffffff, scaled_value_x) * valid;
    data.x = scaled_value_x * inv_scaler;
    int scaled_value_y = static_cast<int>(data.y * scaler);
    scaled_value_y = __reduce_add_sync(0xffffffff, scaled_value_y) * valid;
    data.y = scaled_value_y * inv_scaler;
}

template<>
inline __device__ void warp_reduce_sum<float3, false>(float3& data)
{
    int exponent = (__float_as_uint(data.x) >> 23) & 0xff;
    exponent = max(exponent, (__float_as_uint(data.y) >> 23) & 0xff);
    exponent = max(exponent, (__float_as_uint(data.z) >> 23) & 0xff);
    exponent = __reduce_max_sync(0xffffffff, exponent) - 127;
    int scale_exponent = 23 - exponent;
    bool valid = (exponent > -127) && (scale_exponent < 128);

    float scaler = __uint_as_float(0 | ((scale_exponent + 127) << 23));
    float inv_scaler = __uint_as_float(0 | ((127 - scale_exponent) << 23));

    int scaled_value_x = static_cast<int>(data.x * scaler);
    scaled_value_x = __reduce_add_sync(0xffffffff, scaled_value_x) * valid;
    data.x = scaled_value_x * inv_scaler;
    int scaled_value_y = static_cast<int>(data.y * scaler);
    scaled_value_y = __reduce_add_sync(0xffffffff, scaled_value_y) * valid;
    data.y = scaled_value_y * inv_scaler;
    int scaled_value_z = static_cast<int>(data.z * scaler);
    scaled_value_z = __reduce_add_sync(0xffffffff, scaled_value_z) * valid;
    data.z = scaled_value_z * inv_scaler;
}

template <int tile_size_y, int tile_size_x, bool enable_trans_grad, bool enable_depth_grad>
__global__ void raster_backward_kernel(
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> sorted_points,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> start_index,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> packed_params,         // //[batch,point_num,6]
    const torch::PackedTensorAccessor32<torch::Half, 3, torch::RestrictPtrTraits> packed_rgba16,         // //[batch,point_num,6]
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> specific_tiles,          //[batch,tiles_num]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> final_transmitance,    //[batch,1,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<short, 4, torch::RestrictPtrTraits> last_contributor,    //[batch,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_img,    //[batch,3,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_trans_img,    //[batch,1,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_depth_img,    //[batch,1,tile,tilesize,tilesize]
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> d_ndc,         //[batch,3,point_num]
    torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> d_cov2d_inv,      //[batch,2,2,point_num]
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> d_color,          //[batch,3,point_num]
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> d_opacity,          //[1,point_num]
    int tiles_num_x, int img_h, int img_w)
{
    constexpr int VECTOR_SIZE = 2;
    constexpr int PIXELS_PER_THREAD = (tile_size_x * tile_size_y) / (32 * VECTOR_SIZE);//half2: 32 pixel per warp->64 pixel per warp
    constexpr float SCALER = 128.0f;
    constexpr float INV_SCALER = 1.0f / 128;

    __shared__ half2 shared_img_grad[3][PIXELS_PER_THREAD][4 * 32];
    __shared__ unsigned int shared_last_contributor[PIXELS_PER_THREAD][4 * 32];//ushort2

    const int batch_id = blockIdx.y;
    int tile_id = blockIdx.x * blockDim.y + threadIdx.y + 1;// +1, tile_id 0 is invalid
    if (specific_tiles.size(1) != 0 && (blockIdx.x * blockDim.y + threadIdx.y < specific_tiles.size(1)))
    {
        tile_id = specific_tiles[batch_id][blockIdx.x * blockDim.y + threadIdx.y];
    }

    if (tile_id != 0 && tile_id < start_index.size(1) - 1)
    {

        int start_index_in_tile = start_index[batch_id][tile_id];
        int index_in_tile = 0;

        if (start_index_in_tile != -1)
        {
            BackwardRegisterBuffer reg_buffer[PIXELS_PER_THREAD];
            //int lst[pixels_per_thread];
            #pragma unroll
            for (int i = 0; i < PIXELS_PER_THREAD; i++)
            {
                reg_buffer[i].r = half2(0.0f, 0.0f);
                reg_buffer[i].g = half2(0.0f, 0.0f);
                reg_buffer[i].b = half2(0.0f, 0.0f);

                const int in_tile_x = (threadIdx.x * VECTOR_SIZE) % tile_size_x;
                const int in_tile_y = (threadIdx.x * VECTOR_SIZE) / tile_size_x * PIXELS_PER_THREAD;
                float t0 = final_transmitance[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x];
                float t1 = final_transmitance[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x+1];
                reg_buffer[i].t = half2(t0 * SCALER, t1 * SCALER);

                shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x] = half2(
                    d_img[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x],
                    d_img[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x + 1]);
                shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x] = half2(
                    d_img[batch_id][1][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x],
                    d_img[batch_id][1][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x + 1]); 
                shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x] = half2(
                    d_img[batch_id][2][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x],
                    d_img[batch_id][2][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x + 1]); 
                
                int last0 = last_contributor[batch_id][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x] - 1;
                int last1 = last_contributor[batch_id][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x + 1] - 1;
                index_in_tile = max(max(index_in_tile, last0), last1);
                shared_last_contributor[i][threadIdx.y * blockDim.x + threadIdx.x] = (last1 << 16 | last0);
            }
            index_in_tile = __reduce_max_sync(0xffffffff, index_in_tile);

            const int* points_in_tile = &sorted_points[batch_id][start_index_in_tile];
            const int pixel_x = ((tile_id - 1) % tiles_num_x) * tile_size_x + (threadIdx.x * VECTOR_SIZE) % tile_size_x;
            const int pixel_y = ((tile_id - 1) / tiles_num_x) * tile_size_y + (threadIdx.x * VECTOR_SIZE) / tile_size_x * PIXELS_PER_THREAD;

            for (; (index_in_tile >= 0); index_in_tile--)
            {
                float2 basic;
                float2 bxcy;
                float2 neg_half_c;
                float2 d{ 0,0 };
                int point_id = points_in_tile[index_in_tile];
                PackedParams params = *((PackedParams*)&packed_params[batch_id][point_id][0]);
                {
                    float2 xy{ (float(params.ndc_x) + 1.0f) * 0.5f * img_w - 0.5f ,(float(params.ndc_y) + 1.0f) * 0.5f * img_h - 0.5f };
                    d.x = xy.x - pixel_x;
                    d.y = xy.y - pixel_y;
                    basic=float2{
                    -0.5f * (params.inv_cov00 * d.x * d.x + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * d.x * d.y),
                    -0.5f * (params.inv_cov00 * (d.x - 1) * (d.x - 1) + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * (d.x - 1) * d.y)
                    };
                    bxcy=float2{
                        params.inv_cov11 * d.y + params.inv_cov01 * d.x,
                        params.inv_cov11 * d.y + params.inv_cov01 * (d.x - 1)
                    };
                    neg_half_c=float2{
                        -0.5f * params.inv_cov11,
                        -0.5f * params.inv_cov11
                    };
                }//basic+=(cy+bx)*delta - 0.5*c*delta*delta

                RGBA16 temp = *((RGBA16*)&packed_rgba16[batch_id][point_id][0]);
                RGBA16x2 point_color_x2;
                point_color_x2.r = half2(temp.r, temp.r);
                point_color_x2.g = half2(temp.g, temp.g);
                point_color_x2.b = half2(temp.b, temp.b);
                point_color_x2.a = half2(temp.a, temp.a);
                

                half2 grad_r = half2(0, 0);
                half2 grad_g = half2(0, 0);
                half2 grad_b = half2(0, 0);
                half2 grad_a = half2(0, 0);
                half2 grad_bxcy = half2(0, 0);
                half2 grad_neg_half_c = half2(0, 0);
                half2 grad_basic = half2(0, 0);
                #pragma unroll
                for (int i = 0; i < PIXELS_PER_THREAD; i++)
                {
                    half2 power{ basic.x + i * bxcy.x + i * i * neg_half_c.x,
                        basic.y + i * bxcy.y + i * i * neg_half_c.y };
                    half2 G = fast_exp_approx(power);
                    half2 alpha = point_color_x2.a * G;
                    alpha = __hmin2(half2(255.0f / 256, 255.0f / 256), alpha);

                    unsigned int valid_mask = 0xffffffffu;
                    valid_mask &= __hle2_mask(power, half2(0, 0));
                    valid_mask &= __hge2_mask(alpha, half2(1.0f / 256, 1.0f / 256));
                    valid_mask &= __vcmpleu2(index_in_tile << 16 | index_in_tile, shared_last_contributor[i][threadIdx.y * blockDim.x + threadIdx.x]);

                    if (__any_sync(0xffffffff, valid_mask!=0))
                    {
                        reinterpret_cast<unsigned int*>(&alpha)[0] &= valid_mask;
                        reinterpret_cast<unsigned int*>(&G)[0] &= valid_mask;

                        reg_buffer[i].t = __h2div(reg_buffer[i].t,(half2(1.0f,1.0f) - alpha));//0-2^(-10)
                        grad_r += alpha * reg_buffer[i].t * shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x];
                        grad_g += alpha * reg_buffer[i].t * shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x];
                        grad_b += alpha * reg_buffer[i].t * shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x];

                        half2 d_alpha = half2(0,0);
                        d_alpha += (point_color_x2.r - reg_buffer[i].r) * reg_buffer[i].t * shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x];
                        d_alpha += (point_color_x2.g - reg_buffer[i].g) * reg_buffer[i].t * shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x];
                        d_alpha += (point_color_x2.b - reg_buffer[i].b) * reg_buffer[i].t * shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x];
                        reg_buffer[i].r += alpha * (point_color_x2.r - reg_buffer[i].r);//0-256
                        reg_buffer[i].g += alpha * (point_color_x2.g - reg_buffer[i].g);
                        reg_buffer[i].b += alpha * (point_color_x2.b - reg_buffer[i].b);
                        if (enable_trans_grad)
                        {
                            //d_alpha -= dL_drbgaimg.z * final_transmitance[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x] / (1 - alpha);
                        }

                        grad_a += d_alpha * G;
                        half2 d_G = point_color_x2.a * d_alpha;
                        half2 d_power = G * d_G;
                        grad_bxcy += d_power * half2(i,i);
                        grad_neg_half_c += d_power * half2(i, i) * half2(i, i);
                        grad_basic += d_power;
                    }
                }
                
                //unsigned mask = __ballot_sync(0xffffffff, grad_opacity!=0);
                if (__any_sync(0xffffffff, grad_a.x!=half(0)|| grad_a.y!=half(0)))
                {
                    half2 rg{ grad_r.x + grad_r.y ,grad_g.x + grad_g.y };
                    half2 ba{ grad_b.x + grad_b.y ,grad_a.x + grad_a.y };
                    warp_reduce_sum<half2, false>(rg);
                    warp_reduce_sum<half2, false>(ba);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_color[batch_id][0][point_id], float(rg.x)* INV_SCALER);
                        atomicAdd(&d_color[batch_id][1][point_id], float(rg.y)* INV_SCALER);
                        atomicAdd(&d_color[batch_id][2][point_id], float(ba.x)* INV_SCALER);
                        atomicAdd(&d_opacity[0][point_id], float(ba.y)* INV_SCALER);
                    }

                    float3 grad_invcov{ 0,0,0 };
                    //basic = -0.5f * (params.inv_cov00 * d.x * d.x + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * d.x * d.y);
                    //bxcy = params.inv_cov11 * d.y + params.inv_cov01 * d.x;
                    //neg_half_c = -0.5f * params.inv_cov11;
                    grad_invcov.x = -0.5f * d.x * d.x * float(grad_basic.x);
                    grad_invcov.x += -0.5f * (d.x-1) * (d.x-1) * float(grad_basic.y);
                    grad_invcov.x *= INV_SCALER;
                    grad_invcov.y = (-d.x * d.y * float(grad_basic.x) + d.x * float(grad_bxcy.x)) * 0.5f;
                    grad_invcov.y += (-(d.x-1) * d.y * float(grad_basic.y) + (d.x-1) * float(grad_bxcy.y)) * 0.5f;
                    grad_invcov.y *= INV_SCALER;
                    grad_invcov.z = -0.5f * d.y * d.y * float(grad_basic.x) + d.y * float(grad_bxcy.x) - 0.5f * float(grad_neg_half_c.x);
                    grad_invcov.z += -0.5f * d.y * d.y * float(grad_basic.y) + d.y * float(grad_bxcy.y) - 0.5f * float(grad_neg_half_c.y);
                    grad_invcov.z *= INV_SCALER;

                    warp_reduce_sum<float, false>(grad_invcov.x);
                    warp_reduce_sum<float, false>(grad_invcov.y);
                    warp_reduce_sum<float, false>(grad_invcov.z);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_cov2d_inv[batch_id][0][0][point_id], grad_invcov.x);
                        atomicAdd(&d_cov2d_inv[batch_id][0][1][point_id], grad_invcov.y);
                        atomicAdd(&d_cov2d_inv[batch_id][1][0][point_id], grad_invcov.y);
                        atomicAdd(&d_cov2d_inv[batch_id][1][1][point_id], grad_invcov.z);
                    }

                    float d_dx = (-params.inv_cov00 * d.x - params.inv_cov01 * d.y) * float(grad_basic.x) + params.inv_cov01 * float(grad_bxcy.x);
                    d_dx += (-params.inv_cov00 * (d.x-1) - params.inv_cov01 * d.y) * float(grad_basic.y) + params.inv_cov01 * float(grad_bxcy.y);
                    d_dx *= INV_SCALER;
                    float d_dy = (-params.inv_cov11 * d.y - params.inv_cov01 * d.x) * float(grad_basic.x) + params.inv_cov11 * float(grad_bxcy.x);
                    d_dy += (-params.inv_cov11 * d.y - params.inv_cov01 * (d.x-1)) * float(grad_basic.y) + params.inv_cov11 * float(grad_bxcy.y);
                    d_dy *= INV_SCALER;
                    float2 d_ndc_xy{ d_dx * 0.5f * img_w,d_dy * 0.5f * img_h };
                    warp_reduce_sum<float2, false>(d_ndc_xy);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_ndc[batch_id][0][point_id], d_ndc_xy.x);
                        atomicAdd(&d_ndc[batch_id][1][point_id], d_ndc_xy.y);
                    }
                }
            }
        }
    }
}

template <int tile_size_y, int tile_size_x, bool enable_trans_grad, bool enable_depth_grad>
__global__ void float_raster_backward_kernel(
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> sorted_points,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> start_index,    //[batch,tile]  p.s. tile_id 0 is invalid!
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> packed_params,         // //[batch,point_num,6]
    const torch::PackedTensorAccessor32<torch::Half, 3, torch::RestrictPtrTraits> packed_rgba16,         // //[batch,point_num,4]
    const torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> specific_tiles,          //[batch,tiles_num]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> final_transmitance,    //[batch,1,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<short, 4, torch::RestrictPtrTraits> last_contributor,    //[batch,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_img,    //[batch,3,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_trans_img,    //[batch,1,tile,tilesize,tilesize]
    const torch::PackedTensorAccessor32<float, 5, torch::RestrictPtrTraits> d_depth_img,    //[batch,1,tile,tilesize,tilesize]
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> d_ndc,         //[batch,3,point_num]
    torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> d_cov2d_inv,      //[batch,2,2,point_num]
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> d_color,          //[batch,3,point_num]
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> d_opacity,          //[1,point_num]
    int tiles_num_x, int img_h, int img_w)
{
    constexpr int pixels_per_thread = tile_size_x * tile_size_y / 32;
    __shared__ float shared_img_grad[3][pixels_per_thread][4 * 32];
    __shared__ int shared_last_contributor[pixels_per_thread][4 * 32];

    const int batch_id = blockIdx.y;
    int tile_id = blockIdx.x * blockDim.y + threadIdx.y + 1;// +1, tile_id 0 is invalid
    if (specific_tiles.size(1) != 0 && (blockIdx.x * blockDim.y + threadIdx.y < specific_tiles.size(1)))
    {
        tile_id = specific_tiles[batch_id][blockIdx.x * blockDim.y + threadIdx.y];
    }

    if (tile_id != 0 && tile_id < start_index.size(1) - 1)
    {

        int start_index_in_tile = start_index[batch_id][tile_id];
        int index_in_tile = 0;

        if (start_index_in_tile != -1)
        {
            float4 rgba_buffer[pixels_per_thread];
            //int lst[pixels_per_thread];
#pragma unroll
            for (int i = 0; i < pixels_per_thread; i++)
            {
                const int in_tile_x = threadIdx.x % tile_size_x;
                const int in_tile_y = threadIdx.x / tile_size_x * pixels_per_thread;
                float t = final_transmitance[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x];
                rgba_buffer[i] = float4{ 0.0f,0.0f,0.0f,t };
                shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x] = d_img[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x];
                shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x] = d_img[batch_id][1][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x];
                shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x] = d_img[batch_id][2][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x];
                int lst = last_contributor[batch_id][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x]-1;
                index_in_tile = max(index_in_tile, lst);
                shared_last_contributor[i][threadIdx.y * blockDim.x + threadIdx.x] = lst;
            }
            index_in_tile = __reduce_max_sync(0xffffffff, index_in_tile);

            const int pixel_x = ((tile_id - 1) % tiles_num_x) * tile_size_x + threadIdx.x % tile_size_x;
            const int pixel_y = ((tile_id - 1) / tiles_num_x) * tile_size_y + threadIdx.x / tile_size_x * pixels_per_thread;
            auto points_in_tile = &sorted_points[batch_id][start_index_in_tile];
            for (; (index_in_tile >= 0); index_in_tile--)
            {
                float basic = 0;
                float bxcy = 0;
                float neg_half_c = 0;
                float2 d{ 0,0 };
                int point_id = points_in_tile[index_in_tile];
                PackedParams params = *((PackedParams*)&packed_params[batch_id][point_id][0]);
                {
                    float2 xy{ (float(params.ndc_x) + 1.0f) * 0.5f * img_w - 0.5f ,(float(params.ndc_y) + 1.0f) * 0.5f * img_h - 0.5f };

                    d.x = xy.x - pixel_x;
                    d.y = xy.y - pixel_y;
                    basic = -0.5f * (params.inv_cov00 * d.x * d.x + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * d.x * d.y);
                    bxcy = params.inv_cov11 * d.y + params.inv_cov01 * d.x;
                    neg_half_c = -0.5f * params.inv_cov11;
                }
                //basic+=(cy+bx)*delta - 0.5*c*delta*delta
                RGBA16 temp=*((RGBA16*)&packed_rgba16[batch_id][point_id][0]);
                RGBA32 point_color;
                point_color.r = temp.r;
                point_color.g = temp.g;
                point_color.b = temp.b;
                point_color.a = temp.a;

                float3 grad_color = { 0,0,0 };
                float grad_opacity = 0;
                float grad_bxcy = 0;
                float grad_neg_half_c = 0;
                float grad_basic = 0;
#pragma unroll
                for (int i = 0; i < pixels_per_thread; i++)
                {
                    float power = basic + i * bxcy + i * i * neg_half_c;
                    power = power > 0 ? -6.0f : power;
                    float G = __expf(power);
                    float alpha = min(255.0f / 256, point_color.a * G);
                    bool valid = (index_in_tile <= shared_last_contributor[i][threadIdx.y * blockDim.x + threadIdx.x])
                        && (alpha >= (1.0f / 256));
                    if (__any_sync(0xffffffff, valid))
                    {
                        alpha = valid ? alpha : 0;//0-2^-8
                        G = valid ? G : 0;

                        rgba_buffer[i].w = __fdividef(rgba_buffer[i].w, (1 - alpha));//0-2^(-10)
                        grad_color.x += alpha * rgba_buffer[i].w * shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x];
                        grad_color.y += alpha * rgba_buffer[i].w * shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x];
                        grad_color.z += alpha * rgba_buffer[i].w * shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x];

                        float d_alpha = 0;
                        d_alpha += (point_color.r - rgba_buffer[i].x) * rgba_buffer[i].w * shared_img_grad[0][i][threadIdx.y * blockDim.x + threadIdx.x];
                        d_alpha += (point_color.g - rgba_buffer[i].y) * rgba_buffer[i].w * shared_img_grad[1][i][threadIdx.y * blockDim.x + threadIdx.x];
                        d_alpha += (point_color.b - rgba_buffer[i].z) * rgba_buffer[i].w * shared_img_grad[2][i][threadIdx.y * blockDim.x + threadIdx.x];
                        rgba_buffer[i].x += alpha * (point_color.r - rgba_buffer[i].x);//0-256
                        rgba_buffer[i].y += alpha * (point_color.g - rgba_buffer[i].y);
                        rgba_buffer[i].z += alpha * (point_color.b - rgba_buffer[i].z);
                        if (enable_trans_grad)
                        {
                            //d_alpha -= dL_drbgaimg.z * final_transmitance[batch_id][0][blockIdx.x * blockDim.y + threadIdx.y][in_tile_y + i][in_tile_x] / (1 - alpha);
                        }

                        grad_opacity += d_alpha * G;
                        float d_G = point_color.a * d_alpha;
                        float d_power = G * d_G;
                        grad_bxcy += d_power * i;
                        grad_neg_half_c += d_power * i * i;
                        grad_basic += d_power;
                    }
                }

                //unsigned mask = __ballot_sync(0xffffffff, grad_opacity!=0);
                if (__any_sync(0xffffffff, grad_opacity != 0))
                {
                    half2 rg{ grad_color.x ,grad_color.y };
                    half2 ba{ grad_color.z ,grad_opacity };
                    warp_reduce_sum<half2, false>(rg);
                    warp_reduce_sum<half2, false>(ba);
                    //warp_reduce_sum<float3, false>(grad_color);
                    warp_reduce_sum<float, false>(grad_opacity);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_color[batch_id][0][point_id], float(rg.x));
                        atomicAdd(&d_color[batch_id][1][point_id], float(rg.y));
                        atomicAdd(&d_color[batch_id][2][point_id], float(ba.x));
                        //atomicAdd(&d_color[batch_id][0][point_id], grad_color.x);
                        //atomicAdd(&d_color[batch_id][0][point_id], grad_color.y);
                        //atomicAdd(&d_color[batch_id][0][point_id], grad_color.z);
                        atomicAdd(&d_opacity[0][point_id], grad_opacity);
                    }

                    float3 grad_invcov{ 0,0,0 };
                    //basic = -0.5f * (params.inv_cov00 * d.x * d.x + params.inv_cov11 * d.y * d.y + 2 * params.inv_cov01 * d.x * d.y);
                    //bxcy = params.inv_cov11 * d.y + params.inv_cov01 * d.x;
                    //neg_half_c = -0.5f * params.inv_cov11;
                    grad_invcov.x = -0.5f * d.x * d.x * grad_basic;
                    grad_invcov.y = (-d.x * d.y * grad_basic + d.x * grad_bxcy) * 0.5f;
                    grad_invcov.z = -0.5f * d.y * d.y * grad_basic + d.y * grad_bxcy - 0.5f * grad_neg_half_c;
                    warp_reduce_sum<float, false>(grad_invcov.x);
                    warp_reduce_sum<float, false>(grad_invcov.y);
                    warp_reduce_sum<float, false>(grad_invcov.z);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_cov2d_inv[batch_id][0][0][point_id], grad_invcov.x);
                        atomicAdd(&d_cov2d_inv[batch_id][0][1][point_id], grad_invcov.y);
                        atomicAdd(&d_cov2d_inv[batch_id][1][0][point_id], grad_invcov.y);
                        atomicAdd(&d_cov2d_inv[batch_id][1][1][point_id], grad_invcov.z);
                    }

                    float d_dx = (-params.inv_cov00 * d.x - params.inv_cov01 * d.y) * grad_basic + params.inv_cov01 * grad_bxcy;
                    float d_dy = (-params.inv_cov11 * d.y - params.inv_cov01 * d.x) * grad_basic + params.inv_cov11 * grad_bxcy;
                    float2 d_ndc_xy{ d_dx * 0.5f * img_w,d_dy * 0.5f * img_h };
                    warp_reduce_sum<float2, false>(d_ndc_xy);
                    if (threadIdx.x == 0)
                    {
                        atomicAdd(&d_ndc[batch_id][0][point_id], d_ndc_xy.x);
                        atomicAdd(&d_ndc[batch_id][1][point_id], d_ndc_xy.y);
                    }
                }
            }
        }
    }
}

std::vector<at::Tensor> rasterize_backward(
    at::Tensor sorted_points,
    at::Tensor start_index,
    at::Tensor packed_params,// 
    at::Tensor packed_rgba16,// 
    at::Tensor ndc,// 
    at::Tensor cov2d_inv,
    at::Tensor color,
    at::Tensor opacity,
    std::optional<at::Tensor> specific_tiles_arg,
    at::Tensor final_transmitance,
    at::Tensor last_contributor,
    at::Tensor d_img,
    std::optional<at::Tensor> d_trans_img_arg,
    std::optional<at::Tensor> d_depth_img_arg,
    int64_t img_h,
    int64_t img_w,
    int64_t tilesize_h,
    int64_t tilesize_w
)
{
    at::DeviceGuard guard(packed_params.device());

    int64_t viewsnum = start_index.sizes()[0];
    int tilesnum_x = std::ceil(img_w / float(tilesize_w));
    int tilesnum_y = std::ceil(img_h / float(tilesize_h));
    int64_t tilesnum = tilesnum_x * tilesnum_y;
    at::Tensor specific_tiles;
    if (specific_tiles_arg.has_value())
    {
        specific_tiles = *specific_tiles_arg;
        tilesnum = specific_tiles.sizes()[1];
    }
    else
    {
        specific_tiles = torch::empty({ 0,0 }, packed_params.options().dtype(torch::kInt32));
    }
    at::Tensor d_trans_img;
    if (d_trans_img_arg.has_value())
    {
        d_trans_img = *d_trans_img_arg;
    }
    else
    {
        d_trans_img = torch::empty({ 0,0,0,0,0 }, d_img.options());
    }
    at::Tensor d_depth_img;
    if (d_depth_img_arg.has_value())
    {
        d_depth_img = *d_depth_img_arg;
    }
    else
    {
        d_depth_img = torch::empty({ 0,0,0,0,0 }, d_img.options());
    }
    int batch_num = packed_params.size(0);
    int points_num = packed_params.size(1);
    at::Tensor d_ndc = torch::zeros({ batch_num,4,points_num }, packed_params.options());
    at::Tensor d_cov2d_inv = torch::zeros({ batch_num,2,2,points_num }, packed_params.options());
    at::Tensor d_color = torch::zeros({ batch_num,3,points_num }, packed_params.options());
    at::Tensor d_opacity = torch::zeros({ 1,points_num }, packed_params.options());

    
    int tiles_per_block = 4;
    dim3 Block3d(std::ceil(tilesnum / float(tiles_per_block)), viewsnum, 1);
    dim3 Thread3d(32, tiles_per_block);
    //dim3 Block3d(1, viewsnum, 1);
    //dim3 Thread3d(32, 1);
    raster_backward_kernel<8, 8, false, false> << <Block3d, Thread3d >> > (
        sorted_points.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        start_index.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        packed_params.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        packed_rgba16.packed_accessor32<torch::Half, 3, torch::RestrictPtrTraits>(),
        specific_tiles.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        final_transmitance.packed_accessor32<float, 5, torch::RestrictPtrTraits >(),
        last_contributor.packed_accessor32<short, 4, torch::RestrictPtrTraits>(),
        d_img.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        d_trans_img.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        d_depth_img.packed_accessor32<float, 5, torch::RestrictPtrTraits>(),
        d_ndc.packed_accessor32<float, 3, torch::RestrictPtrTraits >(),
        d_cov2d_inv.packed_accessor32<float, 4, torch::RestrictPtrTraits >(),
        d_color.packed_accessor32<float, 3, torch::RestrictPtrTraits >(),
        d_opacity.packed_accessor32<float, 2, torch::RestrictPtrTraits >(),
        tilesnum_x, img_h, img_w
    );

    CUDA_CHECK_ERRORS;
    return { d_ndc ,d_cov2d_inv ,d_color,d_opacity };
}
